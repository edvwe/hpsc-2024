#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

const int N = 50;
const int RANGE = 5;

// Kernel 1: Zero out the bucket
__global__ void init_bucket(int* bucket) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < RANGE) {
        bucket[i] = 0;
    }
}

// Kernel 2: Count frequency of each value
__global__ void count_instances(int* key, int* bucket, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        atomicAdd(&bucket[key[i]], 1);
    }
}

// Kernel 3: Write sorted values based on bucket counts and prefix offsets
__global__ void fill_sorted(int* key, int* bucket, int* prefix) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < RANGE) {
        for (int j = 0; j < bucket[i]; j++) {
            key[prefix[i] + j] = i;
        }
    }
}

int main() {
    int* key;
    int* bucket;
    int* prefix;

    // Allocate unified memory
    hipMallocManaged(&key, N * sizeof(int));
    hipMallocManaged(&bucket, RANGE * sizeof(int));
    hipMallocManaged(&prefix, RANGE * sizeof(int));  // for offsets

    // Fill key with random values
    printf("Original: ");
    for (int i = 0; i < N; i++) {
        key[i] = rand() % RANGE;
        printf("%d ", key[i]);
    }
    printf("\n");

    // Kernel 1: Init buckets
    init_bucket<<<1, RANGE>>>(bucket);
    hipDeviceSynchronize();

    // Kernel 2: Count frequencies
    count_instances<<<(N + 31) / 32, 32>>>(key, bucket, N);
    hipDeviceSynchronize();

    // Prefix sum on CPU
    prefix[0] = 0;
    for (int i = 1; i < RANGE; i++) {
        prefix[i] = prefix[i - 1] + bucket[i - 1];
    }

    // Kernel 3: Fill sorted keys
    fill_sorted<<<1, RANGE>>>(key, bucket, prefix);
    hipDeviceSynchronize();

    // Output sorted result
    printf("Sorted:   ");
    for (int i = 0; i < N; i++) {
        printf("%d ", key[i]);
    }
    printf("\n");

    hipFree(key);
    hipFree(bucket);
    hipFree(prefix);
    return 0;
}
